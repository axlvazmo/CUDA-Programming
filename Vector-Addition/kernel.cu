﻿/*
* Assignment 4
* Vector addition using cuda device
* Axel Vazquez Montano
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#define N	1024

__global__ void vectorAddi(int *a, int *b, int *c, int n){
	int i = threadIdx.x; //identifying each thread
	if(i < n){	//making sure that we don't go over the needed threads
		c[i] = a[i] + b[i];
	}
}

int main(){
	int *a, *b, *c;		//initializing vectors
	hipMallocManaged(&a, N * sizeof(int));	//allocationg memory in cuda device
	hipMallocManaged(&b, N * sizeof(int));
	hipMallocManaged(&c, N * sizeof(int));

	for (int i = 0; i < N; ++i) {	//assigning values to vectors
		a[i] = i;	//vector calue will be the same as index value
		b[i] = i;
		c[i] = 0;
	}

	vectorAddi <<<1, N>>> (a, b, c, N);		//specifying launch config for kernel <<<1 thread block, number of threads>>> **Note that the number of threads must equal number of elements in the vector to avoid delay in computation

	hipDeviceSynchronize();	//blocking any opperations until cuda device is done computing

	for (int i = 0; i < 20; ++i) {
		printf("c[%d] = %d\n", i, c[i]);
	}

	hipFree(a); // free memory space of the vectors
	hipFree(b);
	hipFree(c);

	return 0;
}