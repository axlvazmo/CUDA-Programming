﻿/*
* Assignment 6
* Task: Multipy a large vector with a scalar and ad another large vector
* Axel Vazquez Montano
*/

#include "hip/hip_runtime.h"

#include <stdio.h>

#define N	500


__global__ void kernelOpps(int *aHost, int *bHost, int *cHost, int scalar, int n) {
	int i = threadIdx.x;
	if (i < n) {
		cHost[i] = (aHost[i] * scalar) + bHost[i];		//computing output
	}
}

int main() {
	
	const int scalar = 2;
	int aHost[N];	//defining values on host device
	int bHost[N];
	int cHost[N];

	/*Assigning initial values to vectors on host*/
	for (int i = 0; i < N; i++) {
		aHost[i] = i;	//values on the vectors will be the value of their index
		bHost[i] = i;
		cHost[i] = 0;
	}

	/*Declaring pointers to cuda Device*/
	int *aDev, *bDev, *cDev;

	/*Allocating memory on cuda device*/
	hipMalloc(&aDev, N * sizeof(int));
	hipMalloc(&bDev, N * sizeof(int));
	hipMalloc(&cDev, N * sizeof(int));

	/*Copying data from host to cuda device*/
	hipMemcpy(aDev, aHost, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(bDev, bHost, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cDev, cHost, N * sizeof(int), hipMemcpyHostToDevice);

	/*Launching Kernel*/
	kernelOpps <<<1, N>>> (aDev, bDev, cDev, scalar, N);

	hipDeviceSynchronize();	//blocking any opperations until cuda device is done computing

	/*Copying data from cuda device to host*/
	hipMemcpy(cHost, cDev, N * sizeof(int), hipMemcpyDeviceToHost);

	/*Printing first 20 results*/
	for (int i = 0; i < 5; ++i) {
		printf("c[%d] = %i\n", i, cHost[i]);
	}

	/*Free cuda device memory*/
	hipFree(aDev);
	hipFree(bDev);
	hipFree(cDev);

	return 0;
}